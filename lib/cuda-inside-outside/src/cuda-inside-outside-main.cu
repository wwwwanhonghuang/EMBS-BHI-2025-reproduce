#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <yaml-cpp/yaml.h>
#include <string>
#include <memory>
#include <grammar/grammar_parser.hpp>
#include "utils/grammar_loader.cuh"
#include "device_management.cuh"
#include "macros.def"
#include <stdio.h>

// __global__ void initialize_buffers(){
    
// }
// Kernel function to initialize the CKY buffer

__device__ float logsumexpf(float a, float b);

struct AlgorithmContext{
    int S = 0;
    int MAX_SEQ_LEN = 0;
    cuda_gc_managed_pt<float> CKY;
    cuda_gc_managed_pt<float> grammar;
    cuda_gc_managed_pt<int> sequence;
    cuda_gc_managed_pt<float> intermediate_results_buffer;
    std::shared_ptr<CudaGC> cuda_gc;

};


void initialize_buffers(AlgorithmContext context){
    context.cuda_gc->fill(context.CKY, -INFINITY);

}


YAML::Node read_yaml_configuration(const std::string& configuration_file_path){
    try {
        YAML::Node config = YAML::LoadFile(configuration_file_path);
        return config;
    } catch (const YAML::Exception& e) {
        std::cerr << "Error loading YAML file: " << e.what() << std::endl;
        return YAML::Node();
    }
}

__global__ void cky_initialization_kernel(int S, int MAX_SEQ_LEN, 
    __device_pt__ float* cky_ptr, __device_pt__ float* grammar_ptr, __device_pt__ int* sequence){
    // Grid-striding for BOTH sequence position (i) and symbol (s_A)

    for (int s_A = blockIdx.y * blockDim.y + threadIdx.y; 
        s_A < S; 
        s_A += blockDim.y * gridDim.y) {
       for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
            i < MAX_SEQ_LEN; 
            i += blockDim.x * gridDim.x) {

        
           // Grammar access: shape [S, S, S]
           int grammar_idx = s_A * (S + 1) * (S + 1) + sequence[i] * (S + 1);
           float rule_val = grammar_ptr[grammar_idx];
           int cky_idx = s_A * (MAX_SEQ_LEN * MAX_SEQ_LEN) + i * MAX_SEQ_LEN + i;

           printf("Grammar[%d, %d, 0] == %lf, %lf\n",  s_A, sequence[i], rule_val, grammar_ptr[0]);


           // CKY table access: shape [S, MAX_SEQ_LEN, MAX_SEQ_LEN]
           cky_ptr[cky_idx] = rule_val;

        }
   }
}

// Helper function for atomic float max
__device__ void atomicMaxFloat(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
            __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
}


__global__ void cky_reduce_kernel(
    int S, 
    int MAX_SEQ_LEN,
    float* __restrict__ cky_table,
    float* __restrict__ intermediate_buffer
){
    // Parallelize across 3D grid: s_A, i, j
    int s_A = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;
    
    // Boundary checks
    if (s_A >= S || i >= MAX_SEQ_LEN || j >= MAX_SEQ_LEN) 
        return;

    float reduced_val = -INFINITY;
    int base_idx = s_A * S * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j;

    // Each thread reduces across s_B dimension
    for (int s_B = 0; s_B < S; s_B++) {
        int buffer_idx = base_idx + s_B * MAX_SEQ_LEN * MAX_SEQ_LEN;
        reduced_val = logsumexpf(reduced_val, intermediate_buffer[buffer_idx]);
    }

    // Write reduced result to CKY table
    cky_table[s_A * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j] = reduced_val;
}


__device__ float logsumexpf(float a, float b) {
    if (a == -INFINITY) return b;
    if (b == -INFINITY) return a;
    float max_ab = fmaxf(a, b);
    return max_ab + logf(expf(a - max_ab) + expf(b - max_ab));
}
__global__ void cky_span_processing_kernel(
    int span_length, int S, int MAX_SEQ_LEN,
    float* __restrict__ cky,
    float* __restrict__ grammar,
    float* __restrict__ results)
{
    // Parallelize over 3 axes: i, s_A, s_B
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int s_A = blockIdx.y * blockDim.y + threadIdx.y;
    int s_B = blockIdx.z * blockDim.z + threadIdx.z;

    // Boundary checks
    if (i >= MAX_SEQ_LEN - span_length || s_A >= S || s_B >= S) return;

    int j = i + span_length;
    if (j >= MAX_SEQ_LEN) return;

    float total_score = -INFINITY;
    const int grammar_stride = (S + 1) * (S + 1);

    // Prefetch the base grammar rule for epsilon productions
    float epsilon_rule = grammar[s_A * grammar_stride + s_B * (S + 1) + 0];

    // Process all possible splits
    for (int k_offset = 0; k_offset < span_length; k_offset++) {
        int k = i + k_offset;
        if (k > j) continue;
        total_score = -INFINITY;
        float left_score = cky[s_B * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + k];
        
        // Handle epsilon production case
        if (k == j) {
            total_score = logsumexpf(total_score, left_score + epsilon_rule);
        }else{
            // Process all right children
            for (int s_C = 1; s_C < S + 1; s_C++) { // s_C cannot equal to 0 (epsilon). 
                float right_score = cky[s_C * MAX_SEQ_LEN * MAX_SEQ_LEN + (k + 1) * MAX_SEQ_LEN + j];
                float rule = grammar[s_A * grammar_stride + s_B * (S + 1) + s_C];
                total_score = logsumexpf(total_score, left_score + right_score + rule);
            }
        }       

    }
    int index_sA_sB_i_j = s_A * (S + 1) * MAX_SEQ_LEN * MAX_SEQ_LEN + s_B * (MAX_SEQ_LEN * MAX_SEQ_LEN) + i * MAX_SEQ_LEN + j;
    results[index_sA_sB_i_j] = logsumexpf(results[index_sA_sB_i_j], total_score); // [s_A, s_B, i, k]

    // // Write result only if valid
    // if (j < MAX_SEQ_LEN) {
    //     if (i == j)
    //         printf("set CKY[%d, %d, %d] = %lf\n", s_A, i, j, total_score);
    //     results[s_A * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j] = total_score;
    // }
}

void cuda_cky_algorithm(AlgorithmContext context) {
    std::cout << "Begin CKY algorithm..." << std::endl;
    std::cout << "Zero out CKY Buffer..." << std::endl;

    initialize_buffers(context);
    std::cout << "[Completed] Zero out CKY Buffer." << std::endl;

    // Launch the kernel to initialize the CKY table
    const int BLOCK_X = 128;  
    const int BLOCK_Y = 8; 

    // Compute grid size (adjust based on your GPU limits)
    int grid_x = min((context.MAX_SEQ_LEN + BLOCK_X - 1) / BLOCK_X, 65535);
    int grid_y = min((context.S + BLOCK_Y - 1) / BLOCK_Y, 65535);

    dim3 blocks(grid_x, grid_y);
    dim3 threads(BLOCK_X, BLOCK_Y);


    std::cout << "Launch CKY span 1 calcualtion kernel..." <<  std::endl;

    cky_initialization_kernel<<<blocks, threads>>>(context.S, context.MAX_SEQ_LEN, 
        context.CKY.ptr, context.grammar.ptr, context.sequence.ptr);
    

    hipError_t cudaerr = hipPeekAtLastError();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    std::cout << "[Completed] CKY span 1 calcualtion." << std::endl;
    
    /* In the CKY algorithm, tasks with a particular span length represent the largest
       parallelizable units of computation. Therefore, we set the largest grain of
       parallelism to the computation over a specific span length. */
    // dim3 cky_blockDim(64, 4, 4);  // Each block has N x S x S threads
    // dim3 cky_gridDim((context.MAX_SEQ_LEN + 64 - 1) / 64, (context.S + 4 - 1) / 4, (context.S + 4 - 1) / 4); 
    // for(int span_length = 2; span_length < context.MAX_SEQ_LEN; span_length++) {
       
    //     cky_span_processing_kernel<<<cky_gridDim, cky_blockDim>>>(
    //         span_length, context.S, context.MAX_SEQ_LEN, 
    //         context.CKY.ptr, context.grammar.ptr, context.intermediate_results_buffer.ptr);
    //     hipDeviceSynchronize();
    // }
    // cky_reduce_kernel<<<cky_gridDim, cky_blockDim>>>(context.S, context.MAX_SEQ_LEN, context.CKY.ptr, context.intermediate_results_buffer.ptr);
    
    std::cout << "[Completed] CKY Algorithm." << std::endl;

}


int main(int argc, char* argv[]) {
    std::string configuration_file_path = "./configurations/config.yaml";
    AlgorithmContext context;

    std::shared_ptr<CudaGC> cuda_gc = std::shared_ptr<CudaGC>();
    context.cuda_gc = cuda_gc;

    if (argc >= 2) {
        configuration_file_path = std::string(argv[1]);
    }

    YAML::Node config = read_yaml_configuration(configuration_file_path);
    if (config.IsNull()) {
        std::cerr << "Failed to load configuration file!" << std::endl;
        return -1;  // Handle the error
    }
    
    int use_device_id = config["cuda_device"]["use_device_id"].as<int>();
    if(select_cuda_device(use_device_id) == 0){
        std::cout << "Use CUDA Device ID: " << use_device_id << std::endl;
    }else{
        return -1;
    }

    int MAX_SEQ_LEN = config["cky_buffer"]["size"]["max_seq_len"].as<int>();
    const std::string& grammar_file_path =  config["grammar"]["file_path"].as<std::string>();
    std::cout << "grammar file path = " << grammar_file_path << std::endl;
    pcfg* parsed_pcfg = prepare_grammar(grammar_file_path);
    __host_pt__ float* host_grammar_buffer = initialize_grammar_buffer_from_pcfg(parsed_pcfg);
    int S = parsed_pcfg->nonterminate_map.size() + parsed_pcfg->terminate_map.size();


    size_t n_cky_buffer_elements = (S + 1) * MAX_SEQ_LEN * MAX_SEQ_LEN;
    size_t n_grammar_buffer_elements = (S + 1) * (S + 1) * (S + 1);
    size_t n_sequence_buffer_elements = MAX_SEQ_LEN; // A -> B C
    long n_intermediate_results_buffer_elements = (S + 1) * (S + 1) * MAX_SEQ_LEN * MAX_SEQ_LEN; // [A, B, i, j]
    std::cout << MAX_SEQ_LEN << "," << S << ", " << S * S * MAX_SEQ_LEN * MAX_SEQ_LEN  << std::endl;
    cuda_gc_managed_pt<float> d_CKY = cuda_gc->allocate<float>(n_cky_buffer_elements);
    cuda_gc_managed_pt<float> grammar = cuda_gc->allocate<float>(n_grammar_buffer_elements);
    cuda_gc_managed_pt<int> sequence = cuda_gc->allocate<int>(n_sequence_buffer_elements);
    cuda_gc->zerolize(grammar);
    cuda_gc->zerolize(sequence);

    cuda_gc_managed_pt<float> intermediate_results_buffer = cuda_gc->allocate<float>(n_intermediate_results_buffer_elements);
    context.S = S;
    context.MAX_SEQ_LEN = MAX_SEQ_LEN;
    context.CKY = d_CKY;
    context.intermediate_results_buffer = intermediate_results_buffer;
    
    initialize_buffers(context);
    cuda_gc->fill(intermediate_results_buffer, -INFINITY);
    hipDeviceSynchronize();
    __host_pt__ int* host_sequence = new int[MAX_SEQ_LEN];
    
    /* [fish people fish tanks]'s ID sequence == [10 9 10 11] + 1*/
    host_sequence[0] = 11;
    host_sequence[1] = 10;
    host_sequence[2] = 11;
    host_sequence[3] = 12;
    host_sequence[4] = 0;
    host_sequence[5] = 0;
    host_sequence[6] = 0;
    host_sequence[7] = 0;
    host_sequence[8] = 0;
    host_sequence[9] = 0;

    hipMemcpy(grammar.ptr, host_grammar_buffer, n_grammar_buffer_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sequence.ptr, host_sequence, n_sequence_buffer_elements * sizeof(int), hipMemcpyHostToDevice);
    context.sequence = sequence;
    context.grammar = grammar;
    cuda_cky_algorithm(context);

    /* Process data in host. */
    // For demonstration: copy a small part of CKY to the host and print a value
    __host_pt__ float* h_CKY = new float[n_cky_buffer_elements];  // Allocate host memory
    hipMemcpy(h_CKY, d_CKY.ptr, n_cky_buffer_elements * sizeof(float), hipMemcpyDeviceToHost);  // Copy data from device to host

    // Print a value for demonstration (example: CKY[0][0][0])
    for(int s = 0; s < S; s++){
        for(int i = 0; i < 4; i++){
            for(int j = i; j < 4; j++){
                std::cout << "CKY[" << s << "][" << i << "][" << j << "]: " 
                << h_CKY[s * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j] << std::endl;
            }
        }
    }

    // Clean up
    delete[] h_CKY; 
    cuda_gc->deallocate<float>(d_CKY);
    hipDeviceReset();

    return 0;
}
